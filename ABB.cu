#include "hip/hip_runtime.h"
/*
* Copyright (c) 2013 Opposite Renderer
* For the full copyright and license information, please view the LICENSE.txt
* file that was distributed with this source code.
*/

#include <optix_world.h>
#include "ppm.h"

using namespace optix;

rtDeclareVariable(float3, cuboidMin, , );
rtDeclareVariable(float3, cuboidMax, , );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


static __device__ float3 boxnormal(float t)
{
	float3 t0 = (cuboidMin - ray.origin)/ray.direction;
	float3 t1 = (cuboidMax - ray.origin)/ray.direction;
	float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
	float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
	return pos-neg;
}

RT_PROGRAM void intersect(int primIdx)
{
	float3 origin = ray.origin;
	if (cuboidMin.x+0.001< origin.x && origin.x< cuboidMax.x-0.001 &&
		cuboidMin.y+0.001< origin.y && origin.y< cuboidMax.y-0.001 &&
		cuboidMin.z+0.001< origin.z && origin.z< cuboidMax.z-0.001 &&
	 (ray.ray_type == ppass_and_gather_ray_type || ray.ray_type == rtpass_ray_type)
	)
	{
		if (rtPotentialIntersection(0.00011f))
		{
			geometricNormal = -ray.direction;
			shadingNormal = geometricNormal;
			rtReportIntersection(0);
		}
		return;
	}

	float3 t0 = (cuboidMin - ray.origin)/ray.direction;
	float3 t1 = (cuboidMax - ray.origin)/ray.direction;
	float3 Near = fminf(t0, t1);
	float3 Far = fmaxf(t0, t1);
	float tmin = fmaxf( Near );
	float tmax = fminf( Far );

	if(tmin <= tmax) {
		bool check_second = true;
		if( rtPotentialIntersection( tmin ) ) {
			shadingNormal = geometricNormal = boxnormal( tmin );
			if(rtReportIntersection(0))
				check_second = false;
		}
		if(check_second) {
			if( rtPotentialIntersection( tmax ) ) {
				shadingNormal = geometricNormal = boxnormal( tmax );
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void boundingBox(int, float result[6])
{
	result[0] = cuboidMin.x;
	result[1] = cuboidMin.y;
	result[2] = cuboidMin.z;
	result[3] = cuboidMax.x;
	result[4] = cuboidMax.y;
	result[5] = cuboidMax.z;
}
