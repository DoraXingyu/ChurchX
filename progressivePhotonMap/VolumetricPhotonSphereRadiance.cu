#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"

using namespace optix;

rtDeclareVariable(VolumetricRadiancePRD, volRadiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, photonPosition, attribute photonPosition, ); 
rtDeclareVariable(float3, photonPower, attribute photonPower, ); 
rtDeclareVariable(uint, photonId, attribute photonId, ); 
rtDeclareVariable(float, photonDist, attribute dist, );
rtDeclareVariable(float, volumetricRadius, ,); 

RT_PROGRAM void anyHitRadiance()
{
    float t = dot(photonPosition-ray.origin, ray.direction)/100;
    //float3 dist3 = photonPosition - make_float3(343.0f, 548.6f, 227.0f);
    //float _dist = sqrt(dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z);
    float dist = photonDist;
    //rtPrintf("%f %f\n", dist, _dist);
    unsigned int frame = floor(dist * FRAME / TOTAL_DISTANCE);
    rtPrintf("%f\n",  dist);
    if(t < ray.tmax && t > ray.tmin)
    {
        volRadiancePrd.radiance[frame] += (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf));
        //rtPrintf("%f %f %f %f\n", t, volumetricRadius, photonPower.x, (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower.x * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf)));
        volRadiancePrd.numHits++;
    }
    rtIgnoreIntersection();
}