#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"

using namespace optix;

rtDeclareVariable(VolumetricRadiancePRD, volRadiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, photonPosition, attribute photonPosition, ); 
rtDeclareVariable(float3, photonPower, attribute photonPower, ); 
rtDeclareVariable(uint, photonId, attribute photonId, ); 

rtDeclareVariable(float, volumetricRadius, ,); 

RT_PROGRAM void anyHitRadiance()
{
    float t = dot(photonPosition-ray.origin, ray.direction)/100;
    float3 dist3 = photonPosition - make_float3(343.0f, 548.6f, 227.0f);
    float dist = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;
    unsigned int frame = floor(dist * FRAME / TOTAL_DISTANCE);
    if(t < ray.tmax && t > ray.tmin)
    {
        volRadiancePrd.radiance[frame] += (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf));
        //rtPrintf("%f %f %f %f\n", t, volumetricRadius, photonPower.x, (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower.x * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf)));
        volRadiancePrd.numHits++;
    }
    rtIgnoreIntersection();
}