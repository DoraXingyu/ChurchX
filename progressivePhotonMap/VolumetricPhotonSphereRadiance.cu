#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"

using namespace optix;

rtDeclareVariable(VolumetricRadiancePRD, volRadiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, photonPosition, attribute photonPosition, ); 
rtDeclareVariable(float3, photonPower, attribute photonPower, ); 
rtDeclareVariable(uint, photonId, attribute photonId, ); 
rtDeclareVariable(float, photonDist, attribute dist, );
rtDeclareVariable(float, volumetricRadius, ,); 

RT_PROGRAM void anyHitRadiance()
{
    float t = dot(photonPosition-ray.origin, ray.direction)/100;
    //float3 dist3 = photonPosition - make_float3(343.0f, 548.6f, 227.0f);
    //float _dist = sqrt(dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z);
    float dist = photonDist;
    //rtPrintf("%f %f\n", dist, _dist);
    int frame = floor(dist * TOTAL_FRAME / TOTAL_DISTANCE) - (START_SECTION*FRAME);
    //int frame = floor(dist * FRAME / TOTAL_DISTANCE);
    bool flag=true;
    #ifdef PREFIX_SUM
        if (frame<0) frame=0;
        if (frame>=FRAME) flag=false;
    #else
        if (frame<0) flag=false;
        if (frame>=FRAME) flag=false;
    #endif
    //rtPrintf("%f\n",  dist);
    if(t < ray.tmax && t > ray.tmin && flag)
    {
        volRadiancePrd.radiance[frame] += (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf));
		//rtPrintf("%f %f %f %f\n", t, volumetricRadius, photonPower.x, (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower.x * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf)));
        volRadiancePrd.numHits++;
    }
    rtIgnoreIntersection();
}