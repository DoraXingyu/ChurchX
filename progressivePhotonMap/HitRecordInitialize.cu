#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"

using namespace optix;

rtBuffer<HitRecord, 3>           rtpass_output_buffer;
rtDeclareVariable(float,         rtpass_default_radius2, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

RT_PROGRAM void kernel()
{
    HitRecord rec;
    rec.normal=rec.position=make_float3(0.0f);

    rec.flags = PPM_NULL;
    rec.radius2 = rtpass_default_radius2;
    rec.photon_count = 0;
    rec.accum_atten = 0.0f;
    rec.flux = make_float3(0.0f, 0.0f, 0.0f);
    rec.accum_atten = 0.0f;
    rec.volumetricRadiance = make_float3(0.0f);
    rtpass_output_buffer[make_uint3(launchIndex.x, launchIndex.y,0)] = rec;
}