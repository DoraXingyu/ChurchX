#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"

using namespace optix;

rtBuffer<HitRecord, 2>           rtpass_output_buffer;
rtDeclareVariable(float,         rtpass_default_radius2, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

RT_PROGRAM void kernel()
{
    HitRecord rec;
    rec.normal=rec.position=make_float3(0.0f);

    rec.flags = PPM_NULL;
    rec.radius2 = rtpass_default_radius2;
    rec.photon_count = 0;
    rec.accum_atten = 0.0f;
    rec.flux = make_float3(0.0f, 0.0f, 0.0f);
    rec.accum_atten = 0.0f;
    for (int i=0; i<FRAME; ++i)
        rec.volumetricRadiance[i] = make_float3(0.0f);
    rtpass_output_buffer[launchIndex] = rec;
}