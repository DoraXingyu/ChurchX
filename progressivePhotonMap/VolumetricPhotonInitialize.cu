#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"



using namespace optix;

rtBuffer<Photon, 1> volumetricPhotons;
rtDeclareVariable(uint1, launchIndex, rtLaunchIndex, );

RT_PROGRAM void kernel()
{
    Photon photon = Photon(make_float3(0), make_float3(0), make_float3(0), 1);
    photon.numDeposits = 0;
    volumetricPhotons[launchIndex.x] = photon;
}

