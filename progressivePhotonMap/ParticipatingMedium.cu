#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"
#include "path_tracer.h"
#include "ppm.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(rtObject, volumetricPhotonsRoot, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(float, sigma_a, , );
rtDeclareVariable(float, sigma_s, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(HitPRD, hitPrd, rtPayload, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );

rtBuffer<Photon, 1> volumetricPhotons;

RT_PROGRAM void closestHitRadiance()
{
    //rtPrintf("Hello2\n");
    const float sigma_t = sigma_a + sigma_s;
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 hitPoint = ray.origin + tHit*ray.direction;
    bool isHitFromOutside = hitFromOutside(ray.direction, worldShadingNormal);
    double tHitStack = tHit + 0.1 - 0.1; // Important, prevents compiler optimization on variable

    /*OPTIX_DEBUG_PRINT(0, "Hit media (%.2f %.2f %.2f) %s (attn: %.2f %.2f  %.2f)\n", hitPoint.x, hitPoint.y, hitPoint.z, isHitFromOutside ? "outside" : "inside",
        radiancePrd.attenuation.x, radiancePrd.attenuation.y, radiancePrd.attenuation.z);*/

    if(isHitFromOutside)
    {
        float3 attenSaved = hitPrd.attenuation + 0.1 - 0.1; // Important, prevents compiler optimization on variable

        // Send ray through the medium
        Ray newRay(hitPoint, ray.direction, radiance_in_participating_medium, 0.01);
        rtTrace(top_object, newRay, hitPrd);
        float distance = hitPrd.lastTHit;
        float transmittance = exp(-distance*sigma_t);
        VolumetricRadiancePRD volRadiancePrd;
        volRadiancePrd.numHits = 0;
        volRadiancePrd.sigma_t = sigma_t;
        volRadiancePrd.sigma_s = sigma_s;
        for (int i=0; i<FRAME; ++i)
            volRadiancePrd.radiance[i]=make_float3(0.0f);
        float3 tmpdirection = ray.direction;
        // Get volumetric radiance
        Ray ray(hitPoint, tmpdirection, volumetric_radiance, 0.0000001, distance);
        //rtPrintf("Volumetric radiance start at: %f %f %f, direction %f %f %f, distance: %f\n", hitPoint.x, hitPoint.y, hitPoint.z, ray.direction.x, ray.direction.y, ray.direction.z, distance);
        rtTrace(volumetricPhotonsRoot, ray, volRadiancePrd);

        // Multiply existing volumetric transmittance with current transmittance, and add gathered volumetric radiance
        // from this path
        // TODO: XINGYU, Can be optimized here by adding a lazy tag
        for (int i=0; i<FRAME; ++i)
        {
            hitPrd.volumetricRadiance[i] *= transmittance;
            hitPrd.volumetricRadiance[i] += attenSaved * volRadiancePrd.radiance[i];
        }
        hitPrd.attenuation *= transmittance;
        //rtPrintf("%f\n", volRadiancePrd.radiance.x);
        //if (volRadiancePrd.numHits>0) rtPrintf("%f %f %f %d \n", distance , transmittance, hitPrd.volumetricRadiance.x, volRadiancePrd.numHits);
    }
    else
    {
        // We are escaping the boundary of the participating medium, so we'll compute the attenuation and volumetric radiance for the remaining path
        // and deliver it to a parent stack frame.
        Ray newRay = Ray(hitPoint, ray.direction, rtpass_ray_type, 0.01);
        //if (hitPrd.volumetricRadiance.x>0)
          //  rtPrintf("hello\n");
        rtTrace(top_object, newRay, hitPrd);
    }
    hitPrd.lastTHit = tHitStack;
}

static __device__ __inline__ float rnd_float_from_uint2(uint2& prev)
{
    return rnd(prev.x) / 2 + rnd(prev.y) / 2;
}

static __device__ __inline__ float2 rnd_from_uint2(uint2& prev)
{
    return make_float2(rnd(prev.x), rnd(prev.y));
}

static __device__ __inline__ optix::float3 sampleScatterSphere(const optix::float3 rayDirection, const optix::float2& sample)
{
	float x = sample.x;
	float y = sample.y;
	//x /= 3.0;
	//y /= 3.0;
	float z = sqrtf(fmaxf(0.0f, 1.0f - x*x - y*y));

	// Now transform into light space
	float3 U, V, W;
	createONB(rayDirection, U, V, W);
	float3 d = x*U + y*V + z*W;
	return d;
}

/*
//
*/

RT_PROGRAM void closestHitPhoton()
{
    //rtPrintf("Hello1\n");
    const float sigma_t = sigma_a + sigma_s;
    photonPrd.ray_depth++;
    photonPrd.dist+=tHit;

    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    bool hitInside = (dot(worldShadingNormal, ray.direction) > 0);

    // If we hit from the inside with a PHOTON_IN_PARTICIPATING_MEDIUM ray, we have escaped the boundry of the medium.
    // We move the ray just a tad to the outside and continue ray tracing there
    if(hitInside && ray.ray_type == photon_in_participating_medium)
    {
        //OPTIX_DEBUG_PRINT(photonPrd.depth-1, "Hit medium P(%.2f %.2f %.2f) from inside: move past\n", hitPoint.x, hitPoint.y, hitPoint.z);
        Ray newRay = Ray(hitPoint+0.0001*ray.direction, ray.direction, ppass_and_gather_ray_type, 0.001, RT_DEFAULT_MAX);
        rtTrace(top_object, newRay, photonPrd);
        return;
    }

    float sample = rnd_float_from_uint2(photonPrd.sample);

    float scatterLocationT = - logf(1-sample)/sigma_t;
    float3 scatterPosition = hitPoint + scatterLocationT*ray.direction;

    //photonPrd.ray_depth = -1;
    int depth = photonPrd.ray_depth;
    // We need to see if anything obstructs the ray in the interval from the hitpoint to the scatter location.
    // If nothings obstructs then we scatter at eventPosition. Otherwise, the photon continues on its path and we don't do anything
    // when we return to this stack frame. We keep the photonPRD depth on the stack to compare it when the rtTrace returns.

    Ray newRay(hitPoint, ray.direction, photon_in_participating_medium, 0.001, scatterLocationT);
    rtTrace(top_object, newRay, photonPrd);
    //rtPrintf("%d\n",photonPrd.ray_depth);

    // If depth is unmodified, no surface was hit from hitpoint to scatterLocation, so we store it as a scatter event.
    // We also scatter a photon in a new direction sampled by the phase function at this location.

    if(depth == photonPrd.ray_depth)
    {

        photonPrd.dist += scatterLocationT;
        
        const float scatterAlbedo = sigma_s/sigma_t;

        if (sample >= scatterAlbedo)
        {
            return;
        }
        //photonPrd.energy *= make_float3(scatterAlbedo);

        // Store photon at scatter location

        //if(photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted)
        {
            int volumetricPhotonIdx = photonPrd.pm_index % NUM_VOLUMETRIC_PHOTONS;
            volumetricPhotons[volumetricPhotonIdx].power = photonPrd.energy;
            volumetricPhotons[volumetricPhotonIdx].position = scatterPosition;
            volumetricPhotons[volumetricPhotonIdx].dist = photonPrd.dist;
            atomicAdd(&volumetricPhotons[volumetricPhotonIdx].numDeposits, 1);
            //rtPrintf("energy: %f, storeID: %d\n",photonPrd.energy.x, volumetricPhotonIdx);
			//rtPrintf("%f %f %f : %f\n", scatterPosition.x, scatterPosition.y, scatterPosition.z, scatterLocationT);
        }
        //rtPrintf("%d\n", photonPrd.ray_depth);
        // Check if we have gone above max number of photons or stack depth
        if(photonPrd.ray_depth >=  3) //TODO:
        {
            return;
        }

        // Create the scattered ray with a direction given by importance sampling of the phase function
        //rtPrintf("Hello1\n");
		float3 scatterDirection = sampleScatterSphere(ray.direction, rnd_from_uint2(photonPrd.sample));
        float tmp= dot(ray.direction, scatterDirection);

        //OPTIX_DEBUG_PRINT(photonPrd.depth-1, "Not interrupted. Store, scatter P(%.2f %.2f %.2f) D(%.2f %.2f %.2f)\n", scatterPosition.x, scatterPosition.y, scatterPosition.z,
          //                scatterDirection.x, scatterDirection.y, scatterDirection.z);
        Ray scatteredRay(scatterPosition, scatterDirection, photon_in_participating_medium, 0.001, RT_DEFAULT_MAX);
        rtTrace(top_object, scatteredRay, photonPrd);

    }
    else
    {
        //rtPrintf("found surface!\n");
        //OPTIX_DEBUG_PRINT(depth-1, "Found surface in [0,t], no scatter!\n");
    }
}
