#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"
#include "path_tracer.h"
#include "random.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );


//
// Ray generation program
//
rtBuffer<PhotonRecord, 1>        ppass_output_buffer;
rtBuffer<uint2, 2>               photon_rnd_seeds;
rtDeclareVariable(uint,          max_depth, , );
rtDeclareVariable(uint,          max_photon_count, , );
rtDeclareVariable(PPMLight,      light , , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

//
// Closest hit material
//
rtDeclareVariable(float3,  Ks, , );
rtDeclareVariable(float3,  Kd, , );
rtDeclareVariable(float3,  emitted, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PhotonPRD, hit_record, rtPayload, );


static __device__ __inline__ float2 rnd_from_uint2(uint2& prev)
{
	return make_float2(rnd(prev.x), rnd(prev.y));
}

RT_PROGRAM void ppass_closest_hit_transparent()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hit_point = ray.origin + t_hit*ray.direction;
	float3 new_ray_dir;

	float3 U, V, W;
	createONB(ffnormal, U, V, W);
	sampleUnitHemisphere(rnd_from_uint2(hit_record.sample), U, V, W, new_ray_dir);

	optix::Ray new_ray( hit_point, new_ray_dir, ppass_and_gather_ray_type, scene_epsilon );
	rtTrace(top_object, new_ray, hit_record);
}

rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );

RT_PROGRAM void gather_any_hit_glass()
{

}

rtDeclareVariable(HitPRD, hit_prd, rtPayload, );
rtBuffer<HitRecord, 3>           rtpass_output_buffer;

rtTextureSampler<float4, 2>      diffuse_map;
rtDeclareVariable(float, diffuse_map_scale, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );


static __device__ __inline__ float3 exp( const float3& x )
{
	return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

rtTextureSampler<float4, 2> envmap;

RT_PROGRAM void rtpass_closest_hit_glass()
{
	float refraction_index = 1.4f;
	float fresnel_exponent = 3.0f;
	float fresnel_minimum = 0.1f;
	float fresnel_maximum = 1.0f;
	float importance_cutoff = 0.01f;
	float3 extinction = make_float3(1.0f, 1.0f, 1.0f);
	float3 extinction_constant= make_float3(log(extinction.x), log(extinction.y), log(extinction.z));
	float3 refraction_color = make_float3(1.0f, 1.0f, 1.0f);

	float3 direction    = ray.direction;
	float3 origin       = ray.origin;
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal     = faceforward( world_shading_normal, -direction, world_geometric_normal );
	float3 hit_point    = origin + t_hit*direction;
	double tHitStack = t_hit + 0.1 - 0.1; // Important, prevents compiler optimization on variable

	if( fmaxf( Kd ) > 0.0f )
	{
		float theta = atan2f( ray.direction.x, ray.direction.z );
		float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
		float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
		float v     = 0.5f * ( 1.0f + sin(phi) );
		float3 result = make_float3(tex2D(envmap, u, v));

		HitRecord rec = rtpass_output_buffer[make_uint3(launch_index.x,launch_index.y,0)];
		// We hit a diffuse surface; record hit and return
		rec.position = hit_point;
		rec.normal = ffnormal;
		rec.attenuated_Kd = Kd * hit_prd.attenuation;

		rec.flags = PPM_HIT;
		rec.attenuated_Kd *= make_float3(
				tex2D(diffuse_map, texcoord.x * diffuse_map_scale, texcoord.y * diffuse_map_scale)) * 100;
		rec.attenuated_Kd += result*2;
		//rtPrintf("%f %f %f\n", result.x, result.y, result.z);
		//rtPrintf("%f %f %f\n", rec.attenuated_Kd.x, rec.attenuated_Kd.y, rec.attenuated_Kd.z);
		rtpass_output_buffer[make_uint3(launch_index.x,launch_index.y,0)] = rec;
	}
	hit_prd.lastTHit = tHitStack;
}

RT_PROGRAM void any_hit_glass_rt()
{
	//rtIgnoreIntersection();
}

RT_PROGRAM void any_hit_glass_ph()
{
	float3 color = make_float3(tex2D(diffuse_map, texcoord.x*diffuse_map_scale, texcoord.y*diffuse_map_scale));
	//rtPrintf("%f %f %f\n", color.x, color.y, color.z);
	hit_record.energy*=color*5;
	rtIgnoreIntersection();
}