#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"

using namespace optix;

rtDeclareVariable(VolumetricRadiancePRD, volRadiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, photonPosition, attribute photonPosition, ); 
rtDeclareVariable(float3, photonPower, attribute photonPower, ); 
rtDeclareVariable(uint, photonId, attribute photonId, ); 

rtDeclareVariable(float, volumetricRadius, ,); 

RT_PROGRAM void anyHitRadiance()
{
    float t = dot(photonPosition-ray.origin, ray.direction);

    if(t < ray.tmax && t > ray.tmin)
    {
        volRadiancePrd.radiance += (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf));
        //rtPrintf("%f %f %f\n", volumetricRadius, photonPower.x, (1/(M_PIf*volumetricRadius*volumetricRadius)) * photonPower.x * exp(-volRadiancePrd.sigma_t*t) * (1.f/(4.f*M_PIf)));
        volRadiancePrd.numHits++;
    }
    rtIgnoreIntersection();
}