#include "hip/hip_runtime.h"
/*
* Copyright (c) 2013 Opposite Renderer
* For the full copyright and license information, please view the LICENSE.txt
* file that was distributed with this source code.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"
#include "ppm.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(rtObject, volumetricPhotonsRoot, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(float, sigma_a, , );
rtDeclareVariable(float, sigma_s, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(HitPRD, hitPrd, rtPayload, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );

rtBuffer<Photon, 1> volumetricPhotons;

RT_PROGRAM void closestHitRadiance()
{
    const float sigma_t = sigma_a + sigma_s;
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 hitPoint = ray.origin + tHit*ray.direction;
    bool isHitFromOutside = hitFromOutside(ray.direction, worldShadingNormal);
    double tHitStack = tHit + 0.1 - 0.1; // Important, prevents compiler optimization on variable

    /*OPTIX_DEBUG_PRINT(0, "Hit media (%.2f %.2f %.2f) %s (attn: %.2f %.2f  %.2f)\n", hitPoint.x, hitPoint.y, hitPoint.z, isHitFromOutside ? "outside" : "inside",
        radiancePrd.attenuation.x, radiancePrd.attenuation.y, radiancePrd.attenuation.z);*/

    if(isHitFromOutside)
    {
        float3 attenSaved = hitPrd.attenuation + 0.1 - 0.1; // Important, prevents compiler optimization on variable

        // Send ray through the medium
        Ray newRay(hitPoint, ray.direction, radiance_in_participating_medium, 0.01);
        rtTrace(top_object, newRay, hitPrd);

        float distance = hitPrd.lastTHit;
        float transmittance = exp(-distance*sigma_t);

        VolumetricRadiancePRD volRadiancePrd;
        volRadiancePrd.radiance = make_float3(0);
        volRadiancePrd.numHits = 0;
        volRadiancePrd.sigma_t = sigma_t;
        volRadiancePrd.sigma_s = sigma_s;

        // Get volumetric radiance

        Ray ray(hitPoint, ray.direction, volumetric_radiance, 0.0000001, distance);
        rtTrace(volumetricPhotonsRoot, ray, volRadiancePrd);

        // Multiply existing volumetric transmittance with current transmittance, and add gathered volumetric radiance
        // from this path
        float3 tmp = hitPrd.volumetricRadiance;
        hitPrd.volumetricRadiance *= transmittance;
        hitPrd.volumetricRadiance += attenSaved*volRadiancePrd.radiance;
        hitPrd.attenuation *= transmittance;

    }
    else
    {
        // We are escaping the boundary of the participating medium, so we'll compute the attenuation and volumetric radiance for the remaining path
        // and deliver it to a parent stack frame.
        Ray newRay = Ray(hitPoint, ray.direction, rtpass_ray_type, 0.01);
        //if (hitPrd.volumetricRadiance.x>0)
          //  rtPrintf("hello\n");
        rtTrace(top_object, newRay, hitPrd);
    }
    hitPrd.lastTHit = tHitStack;

}

static __device__ __inline__ float rnd_float_from_uint2(uint2& prev)
{
    return rnd(prev.x) / 2 + rnd(prev.y) / 2;
}

static __device__ __inline__ float2 rnd_from_uint2(uint2& prev)
{
    return make_float2(rnd(prev.x), rnd(prev.y));
}

/*
//
*/

RT_PROGRAM void closestHitPhoton()
{
    const float sigma_t = sigma_a + sigma_s;

    photonPrd.ray_depth++;

    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    bool hitInside = (dot(worldShadingNormal, ray.direction) > 0);

    // If we hit from the inside with a PHOTON_IN_PARTICIPATING_MEDIUM ray, we have escaped the boundry of the medium.
    // We move the ray just a tad to the outside and continue ray tracing there
    if(hitInside && ray.ray_type == photon_in_participating_medium)
    {
        //OPTIX_DEBUG_PRINT(photonPrd.depth-1, "Hit medium P(%.2f %.2f %.2f) from inside: move past\n", hitPoint.x, hitPoint.y, hitPoint.z);
        Ray newRay = Ray(hitPoint+0.0001*ray.direction, ray.direction, ppass_and_gather_ray_type, 0.001, RT_DEFAULT_MAX);
        rtTrace(top_object, newRay, photonPrd);
        return;
    }

    float sample = rnd_float_from_uint2(photonPrd.sample);

    float scatterLocationT = - logf(1-sample)/sigma_t;
    float3 scatterPosition = hitPoint + scatterLocationT*ray.direction;
    int depth = photonPrd.ray_depth;

    // We need to see if anything obstructs the ray in the interval from the hitpoint to the scatter location.
    // If nothings obstructs then we scatter at eventPosition. Otherwise, the photon continues on its path and we don't do anything
    // when we return to this stack frame. We keep the photonPRD depth on the stack to compare it when the rtTrace returns.

    Ray newRay(hitPoint, ray.direction, photon_in_participating_medium, 0.001, scatterLocationT);
    rtTrace(top_object, newRay, photonPrd);

    // If depth is unmodified, no surface was hit from hitpoint to scatterLocation, so we store it as a scatter event.
    // We also scatter a photon in a new direction sampled by the phase function at this location.

    if(depth == photonPrd.ray_depth)
    {
        const float scatterAlbedo = sigma_s/sigma_t;

        if (sample >= scatterAlbedo)
        {
            return;
        }
        //photonPrd.power *= scatterAlbedo;

        // Store photon at scatter location

        //if(photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted)
        {
            int volumetricPhotonIdx = photonPrd.pm_index % 200000; //TODO:
            volumetricPhotons[volumetricPhotonIdx].power = photonPrd.energy;
            volumetricPhotons[volumetricPhotonIdx].position = scatterPosition;
            atomicAdd(&volumetricPhotons[volumetricPhotonIdx].numDeposits, 1);
        }

        // Check if we have gone above max number of photons or stack depth
        if(photonPrd.ray_depth >=  15) //TODO:
        {
            return;
        }

        // Create the scattered ray with a direction given by importance sampling of the phase function

        float3 scatterDirection = sampleUnitSphere(rnd_from_uint2(photonPrd.sample));

        //OPTIX_DEBUG_PRINT(photonPrd.depth-1, "Not interrupted. Store, scatter P(%.2f %.2f %.2f) D(%.2f %.2f %.2f)\n", scatterPosition.x, scatterPosition.y, scatterPosition.z,
          //                scatterDirection.x, scatterDirection.y, scatterDirection.z);

        Ray scatteredRay(scatterPosition, scatterDirection, ppass_and_gather_ray_type, 0.001, RT_DEFAULT_MAX);
        rtTrace(top_object, scatteredRay, photonPrd);

    }
    else
    {
        //OPTIX_DEBUG_PRINT(depth-1, "Found surface in [0,t], no scatter!\n");
    }
}
