#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"
#include "path_tracer.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, emitted, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PhotonPRD, hit_record, rtPayload, );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float, scene_epsilon, , );
rtBuffer<PhotonRecord, 1>        ppass_output_buffer;

static __device__ __inline__ float2 rnd_from_uint2(uint2& prev)
{
	return make_float2(rnd(prev.x), rnd(prev.y));
}

RT_PROGRAM void ppass_closest_hit_transparent()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hit_point = ray.origin;
	float3 new_ray_dir;

	float3 U, V, W;
	createONB(ffnormal, U, V, W);
	sampleUnitHemisphere(rnd_from_uint2(hit_record.sample), U, V, W, new_ray_dir);

	optix::Ray new_ray( hit_point, new_ray_dir, ppass_and_gather_ray_type, scene_epsilon );
	rtTrace(top_object, new_ray, hit_record);
}

rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );

RT_PROGRAM void gather_any_hit_glass()
{

}

rtDeclareVariable(HitPRD, hit_prd, rtPayload, );
rtBuffer<HitRecord, 2>           rtpass_output_buffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void rtpass_closest_hit_glass()
{
	float3 hit_point = ray.origin + t_hit*ray.direction;
	float3 new_ray_dir = ray.direction;
	hit_record.energy = hit_record.energy;
	optix::Ray new_ray(hit_point, new_ray_dir, rtpass_ray_type, scene_epsilon);
	rtTrace(top_object, new_ray, hit_record);
}