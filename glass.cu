#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ppm.h"
#include "path_tracer.h"
#include "random.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );


//
// Ray generation program
//
rtBuffer<PhotonRecord, 1>        ppass_output_buffer;
rtBuffer<uint2, 2>               photon_rnd_seeds;
rtDeclareVariable(uint,          max_depth, , );
rtDeclareVariable(uint,          max_photon_count, , );
rtDeclareVariable(PPMLight,      light , , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

//
// Closest hit material
//
rtDeclareVariable(float3,  Ks, , );
rtDeclareVariable(float3,  Kd, , );
rtDeclareVariable(float3,  emitted, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PhotonPRD, hit_record, rtPayload, );


static __device__ __inline__ float2 rnd_from_uint2(uint2& prev)
{
	return make_float2(rnd(prev.x), rnd(prev.y));
}

RT_PROGRAM void ppass_closest_hit_transparent()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hit_point = ray.origin + t_hit*ray.direction;
	float3 new_ray_dir;

	float3 U, V, W;
	createONB(ffnormal, U, V, W);
	sampleUnitHemisphere(rnd_from_uint2(hit_record.sample), U, V, W, new_ray_dir);

	optix::Ray new_ray( hit_point, new_ray_dir, ppass_and_gather_ray_type, scene_epsilon );
	rtTrace(top_object, new_ray, hit_record);
}

rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );

RT_PROGRAM void gather_any_hit_glass()
{

}

rtDeclareVariable(HitPRD, hit_prd, rtPayload, );
rtBuffer<HitRecord, 2>           rtpass_output_buffer;

rtTextureSampler<float4, 2>      diffuse_map;
rtDeclareVariable(float, diffuse_map_scale, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );


static __device__ __inline__ float3 exp( const float3& x )
{
	return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

RT_PROGRAM void rtpass_closest_hit_glass()
{
	float refraction_index = 1.4f;
	float fresnel_exponent = 3.0f;
	float fresnel_minimum = 0.1f;
	float fresnel_maximum = 1.0f;
	float importance_cutoff = 0.01f;
	float3 extinction = make_float3(1.0f, 1.0f, 1.0f);
	float3 extinction_constant= make_float3(log(extinction.x), log(extinction.y), log(extinction.z));
	float3 refraction_color = make_float3(1.0f, 1.0f, 1.0f);

	float3 direction    = ray.direction;
	float3 origin       = ray.origin;
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal     = faceforward( world_shading_normal, -direction, world_geometric_normal );
	float3 hit_point    = origin + t_hit*direction;
	double tHitStack = t_hit + 0.1 - 0.1; // Important, prevents compiler optimization on variable

	if( fmaxf( Kd ) > 0.0f )
	{
		HitRecord rec = rtpass_output_buffer[launch_index];;
		float reflection = 1.0f;
		float3 result = make_float3(0.0f);

		float3 beer_attenuation;
		if(dot(world_shading_normal, ray.direction) > 0){
			// Beer's law attenuation
			beer_attenuation = exp(extinction_constant * 0.01);
		} else {
			beer_attenuation = exp(extinction_constant * 0.01);
		}

		float3 t;                                                            // transmission direction
		if ( refract(t, direction, world_shading_normal, refraction_index) )
		{
			// check for external or internal reflection
			float cos_theta = dot(direction, world_shading_normal);
			if (cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, world_shading_normal);

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			optix::Ray ray(hit_point, t, radiance_in_participating_medium, scene_epsilon);
			HitPRD refr_prd;
			refr_prd.ray_depth = hit_prd.ray_depth + 1;
			refr_prd.attenuation = hit_prd.attenuation;

			rtTrace(top_object, ray, refr_prd);
			result += (1.0f - reflection) * refraction_color * refr_prd.attenuation;
		}

		// We hit a diffuse surface; record hit and return
		rec.position = hit_point;
		rec.normal = ffnormal;
		rec.attenuated_Kd = (Kd + result) * hit_prd.attenuation;

		rec.flags = PPM_HIT;
		rec.attenuated_Kd *= make_float3(
				tex2D(diffuse_map, texcoord.x * diffuse_map_scale, texcoord.y * diffuse_map_scale));
		//rtPrintf("%f %f %f\n", rec.attenuated_Kd.x, rec.attenuated_Kd.y, rec.attenuated_Kd.z);
		rtpass_output_buffer[launch_index] = rec;
	}
	hit_prd.lastTHit = tHitStack;
}

RT_PROGRAM void any_hit_glass_rt()
{
	//rtIgnoreIntersection();
}

RT_PROGRAM void any_hit_glass_ph()
{
	float3 color = make_float3(tex2D(diffuse_map, texcoord.x*diffuse_map_scale, texcoord.y*diffuse_map_scale));
	//rtPrintf("%f %f %f\n", color.x, color.y, color.z);
	hit_record.energy*=color*5;
	rtIgnoreIntersection();
}